#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
// 

__global__ void my_first_kernel(float *x, float *v1, float *v2)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = v1[tid]+v2[tid];

  printf("Thread = %d, x = %f\n", tid, x[tid]);
}


//
// main code
//

int main(int argc, const char **argv)
{
  float *h_x, *d_x;
  float *h_v1, *d_v1, *h_v2, *d_v2;
  int   nblocks, nthreads, nsize, n; 

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h_x = (float *)malloc(nsize*sizeof(float));
  checkCudaErrors(hipMalloc((void **)&d_x, nsize*sizeof(float)));
  h_v1 = (float *)malloc(nsize*sizeof(float));
  checkCudaErrors(hipMalloc((void **)&d_v1, nsize*sizeof(float)));
  h_v2 = (float *)malloc(nsize*sizeof(float));
  checkCudaErrors(hipMalloc((void **)&d_v2, nsize*sizeof(float)));

  // Allocate on host

  for (n=0; n<nsize; n++){
	h_v1[n] = (float) n;
	h_v2[n] = (float) 2*n;
  }

  // Copy to device

checkCudaErrors( hipMemcpy(d_v1,h_v1,nsize*sizeof(float),
                 hipMemcpyHostToDevice) );
checkCudaErrors( hipMemcpy(d_v2,h_v2,nsize*sizeof(float),
                 hipMemcpyHostToDevice) );

  // execute kernel
  
  my_first_kernel<<<nblocks,nthreads>>>(d_x, d_v1, d_v2);
  getLastCudaError("my_first_kernel execution failed\n");

  // copy back results and print them out

  checkCudaErrors( hipMemcpy(h_x,d_x,nsize*sizeof(float),
                 hipMemcpyDeviceToHost) );

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // free memory 

  checkCudaErrors(hipFree(d_x));
  free(h_x);
  checkCudaErrors(hipFree(d_v1));
  free(h_v1);
  checkCudaErrors(hipFree(d_v2));
  free(h_v2);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
