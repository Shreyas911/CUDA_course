#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////////////
// CPU routines
////////////////////////////////////////////////////////////////////////////////

void reduction_gold(float* odata, float* idata, const unsigned int len) 
{
  *odata = 0;
  for(int i=0; i<len; i++) *odata += idata[i];
  printf("CPU sum = %f\n", *odata); 
}

////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////

__global__ void reduction(float *g_idata, float *d_global_sum)
{
    // dynamically allocated shared memory

    extern  __shared__  float temp[];

    int global_tid = threadIdx.x + blockDim.x*blockIdx.x;
    int tid = threadIdx.x;

    // first, each thread loads data into shared memory
    temp[tid] = g_idata[global_tid];

    // next, we perform binary tree reduction

    for (int d = blockDim.x>>1; d > 0; d >>= 1) {
      __syncthreads();  // ensure previous step completed 
      if (tid<d)  temp[tid] += temp[tid+d];
    }

    // finally, first thread puts result into global memory
    if (tid==0) atomicAdd(d_global_sum, temp[0]);
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, const char** argv) 
{
  int num_blocks, num_elements, num_threads, mem_size, shared_mem_size;

  float *h_data, *reference, sum, *h_global_sum;;
  float *d_idata, *d_global_sum;

  // initialise card

  findCudaDevice(argc, argv);

  num_blocks   = 2;
  num_elements = num_blocks*512;
  num_threads  = 512;
  mem_size     = sizeof(float) * num_elements;

  // allocate host memory to store the input data
  // and initialize to integer values between 0 and 1000

  h_data = (float*) malloc(mem_size);
  h_global_sum = (float*) malloc(sizeof(float));
      
  for(int i = 0; i < num_elements; i++) 
    h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));

  // compute reference solutions

  reference = (float*) malloc(mem_size);
  reduction_gold(&sum, h_data, num_elements);

  // allocate device memory input and output arrays

  checkCudaErrors( hipMalloc((void**)&d_idata, mem_size) );
  checkCudaErrors( hipMalloc((void**)&d_global_sum, sizeof(float)) );

 // copy host memory to device input array

  checkCudaErrors( hipMemcpy(d_idata, h_data, mem_size,
                              hipMemcpyHostToDevice) );

  // execute the kernel

  shared_mem_size = sizeof(float) * num_threads;
  reduction<<<num_blocks,num_threads,shared_mem_size>>>(d_idata, d_global_sum);
  hipDeviceSynchronize();
  getLastCudaError("reduction kernel execution failed");

  // copy result from device to host

  checkCudaErrors( hipMemcpy(h_global_sum, d_global_sum, sizeof(float),
                              hipMemcpyDeviceToHost) );

  // check results

  printf("reduction error = %f\n",h_global_sum[0]-sum);

  // cleanup memory

  free(h_data);
  free(h_global_sum);
  free(reference);
  checkCudaErrors( hipFree(d_idata) );
  checkCudaErrors( hipFree(d_global_sum) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
}
