#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////////////
// CPU routines
////////////////////////////////////////////////////////////////////////////////

void reduction_gold(int* odata, int* idata, const unsigned int len) 
{
  *odata = 0;
  for(int i=0; i<len; i++) *odata += idata[i];
  printf("CPU sum = %d\n", *odata); 
}

////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////

__global__ void reduction(int *g_idata, int *d_global_sum)
{
    // dynamically allocated shared memory

    extern  __shared__  int temp[];

    int global_tid = threadIdx.x + blockDim.x*blockIdx.x;
    int tid = threadIdx.x;
    int warp_id = threadIdx.x / 32;
    temp[tid] = g_idata[global_tid];
    int warp_sum;

    warp_sum = __reduce_add_sync(-1, temp[tid]);
    if((tid+1)%32==0) {
      atomicAdd(d_global_sum,warp_sum);
    }
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, const char** argv) 
{
  int num_blocks, num_elements, num_threads, mem_size, shared_mem_size;

  int *h_data, *reference, sum, *h_global_sum;;
  int *d_idata, *d_global_sum;

  // initialise card

  findCudaDevice(argc, argv);

  num_blocks   = 2;
  num_elements = num_blocks*512;
  num_threads  = 512;
  mem_size     = sizeof(int) * num_elements;

  // allocate host memory to store the input data
  // and initialize to integer values between 0 and 1000

  h_data = (int*) malloc(mem_size);
  h_global_sum = (int*) malloc(sizeof(int));
      
  for(int i = 0; i < num_elements; i++) 
    h_data[i] = 1;

  // compute reference solutions

  reference = (int*) malloc(mem_size);
  reduction_gold(&sum, h_data, num_elements);

  // allocate device memory input and output arrays

  checkCudaErrors( hipMalloc((void**)&d_idata, mem_size) );
  checkCudaErrors( hipMalloc((void**)&d_global_sum, sizeof(float)) );

 // copy host memory to device input array

  checkCudaErrors( hipMemcpy(d_idata, h_data, mem_size,
                              hipMemcpyHostToDevice) );

  // execute the kernel

  shared_mem_size = sizeof(int) * num_threads;
  reduction<<<num_blocks,num_threads,shared_mem_size>>>(d_idata, d_global_sum);
  hipDeviceSynchronize();
  getLastCudaError("reduction kernel execution failed");

  // copy result from device to host

  checkCudaErrors( hipMemcpy(h_global_sum, d_global_sum, sizeof(float),
                              hipMemcpyDeviceToHost) );

  // check results

  printf("reduction error = %d\n",h_global_sum[0]-sum);

  // cleanup memory

  free(h_data);
  free(h_global_sum);
  free(reference);
  checkCudaErrors( hipFree(d_idata) );
  checkCudaErrors( hipFree(d_global_sum) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
}
