#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////////////
// CPU routines
////////////////////////////////////////////////////////////////////////////////

void reduction_gold(float* odata, float* idata, const unsigned int len) 
{
  *odata = 0;
  for(int i=0; i<len; i++) *odata += idata[i];
  printf("CPU sum = %f\n", *odata); 
}

////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////

__global__ void reduction(float *g_odata, float *g_idata)
{
    // dynamically allocated shared memory

    extern  __shared__  float temp[];

    int global_tid = threadIdx.x + blockDim.x*blockIdx.x;
    int tid = threadIdx.x;

    // first, each thread loads data into shared memory
    temp[tid] = g_idata[global_tid];

    // next, we perform binary tree reduction

    for (int d = blockDim.x>>1; d > 0; d >>= 1) {
      __syncthreads();  // ensure previous step completed 
      if (tid<d)  temp[tid] += temp[tid+d];
    }

    // finally, first thread puts result into global memory
    if (tid==0) g_odata[blockIdx.x] = temp[0];
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, const char** argv) 
{
  int num_blocks, num_elements, num_threads, mem_size, shared_mem_size;

  float *h_data, *reference, sum, sum_gpu;
  float *d_idata, *d_odata;

  // initialise card

  findCudaDevice(argc, argv);

  num_blocks   = 2;
  num_elements = num_blocks*512;
  num_threads  = 512;
  mem_size     = sizeof(float) * num_elements;

  // allocate host memory to store the input data
  // and initialize to integer values between 0 and 1000

  h_data = (float*) malloc(mem_size);
      
  for(int i = 0; i < num_elements; i++) 
    h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));

  // compute reference solutions

  reference = (float*) malloc(mem_size);
  reduction_gold(&sum, h_data, num_elements);

  // allocate device memory input and output arrays

  checkCudaErrors( hipMalloc((void**)&d_idata, mem_size) );
  checkCudaErrors( hipMalloc((void**)&d_odata, num_blocks*sizeof(float)) );

  // copy host memory to device input array

  checkCudaErrors( hipMemcpy(d_idata, h_data, mem_size,
                              hipMemcpyHostToDevice) );

  // execute the kernel

  shared_mem_size = sizeof(float) * num_threads;
  reduction<<<num_blocks,num_threads,shared_mem_size>>>(d_odata,d_idata);
  hipDeviceSynchronize();
  getLastCudaError("reduction kernel execution failed");

  // copy result from device to host

  checkCudaErrors( hipMemcpy(h_data, d_odata, num_blocks*sizeof(float),
                              hipMemcpyDeviceToHost) );

  // check results

  sum_gpu = 0.0;
  for(int i = 0; i < num_blocks; i++)
    sum_gpu += h_data[i];

  printf("reduction error = %f\n",sum_gpu-sum);

  // cleanup memory

  free(h_data);
  free(reference);
  checkCudaErrors( hipFree(d_idata) );
  checkCudaErrors( hipFree(d_odata) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
}
